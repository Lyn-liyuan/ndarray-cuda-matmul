#include <stdio.h>
#include <stdlib.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include <mutex>
#include <atomic>

static hipblasHandle_t cublas_handle = nullptr;
static std::atomic<int> refCounter(0);
std::mutex mtx;

extern "C" void _init_cublas()
{
    if (cublas_handle == nullptr)
    {
        std::lock_guard<std::mutex> lock(mtx);
        if (cublas_handle == nullptr)
        {
            cublasCheck(hipblasCreate(&cublas_handle));
            cublasCheck(hipblasSetMathMode(cublas_handle, HIPBLAS_TF32_TENSOR_OP_MATH));
        }
    }
    refCounter.fetch_add(1, std::memory_order_relaxed);
}

extern "C" void _destory_cublas()
{
    if (cublas_handle != nullptr && refCounter.load(std::memory_order_relaxed)>0)
    {
        refCounter.fetch_sub(1, std::memory_order_relaxed);
        if (refCounter.load(std::memory_order_relaxed) == 0)
        {
            std::lock_guard<std::mutex> lock(mtx);

            if (cublas_handle != nullptr)
            {
                cublasCheck(hipblasDestroy(cublas_handle));
                cublas_handle = nullptr;
            }
        }
    }
}

extern "C" struct Mat
{
    float * data;
    size_t size;
};

typedef struct Mat Mat;

extern "C" void to_host(float *out , Mat *m) {
    cudaCheck(hipMemcpy(out, m->data, m->size * sizeof(float), hipMemcpyDeviceToHost));
}

extern "C" float * to_device(float *in, size_t size) {
    float *out;
    cudaCheck(hipMalloc(&out, size * sizeof(float)));  // Allocate GPU memory
    cudaCheck(hipMemcpy(out, in, size * sizeof(float), hipMemcpyHostToDevice));  // Corrected size

    return out;
}

extern "C" void cuda_free(float *m) {
    if(m) cudaCheck(hipFree(m));
}

extern "C" void mat_free(Mat * m) {
    if(m) {
        delete m;
    }
}

extern "C" Mat * matmul_cublas_device(const float *a, const float *b,
                              int m, int n, int k)
{
    const float alpha = 1.0f;
    const float beta = 0.0f;
    float * out_mat;
    cudaCheck(hipMalloc(&out_mat, m * n * sizeof(float)));
    cublasCheck(hipblasSgemm(cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_T, m, n, k, &alpha, a, k, b, n, &beta, out_mat, m));
    
    Mat * mat = new Mat();
    mat->data = out_mat;
    mat->size = m * n * sizeof(float);
    return mat;
}

extern "C" void matmul_cublas(float *out,
                              const float *a, const float *b,
                              int m, int n, int k)
{

    const float alpha = 1.0f;
    const float beta = 0.0f;
    float *a_mat, *b_mat, *out_mat;
    cudaCheck(hipMalloc(&a_mat, m * k * sizeof(float)));
    cudaCheck(hipMalloc(&b_mat, n * k * sizeof(float)));
    cudaCheck(hipMalloc(&out_mat, m * n * sizeof(float)));
    cudaCheck(hipMemcpy(a_mat, a, m * k * sizeof(float), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(b_mat, b, n * k * sizeof(float), hipMemcpyHostToDevice));

    cublasCheck(hipblasSgemm(cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_T, m, n, k, &alpha, a_mat, k, b_mat, n, &beta, out_mat, m));
    cudaCheck(hipMemcpy(out, out_mat, m * n * sizeof(float), hipMemcpyDeviceToHost));
    if (a_mat)
        cudaCheck(hipFree(a_mat));
    if (b_mat)
        cudaCheck(hipFree(b_mat));
    if (out_mat)
        cudaCheck(hipFree(out_mat));
}
